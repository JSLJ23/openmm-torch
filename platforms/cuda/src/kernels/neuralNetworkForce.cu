#include "hip/hip_runtime.h"
extern "C" __global__
void addForces(const FORCES_TYPE* __restrict__ grads, long long* __restrict__ forceBuffers, int* __restrict__ atomIndex, int numAtoms, int paddedNumAtoms) {
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < numAtoms; atom += blockDim.x*gridDim.x) {
        int index = atomIndex[atom];
        forceBuffers[atom] -= (long long) (grads[3*index]*0x100000000);
        forceBuffers[atom+paddedNumAtoms] -= (long long) (grads[3*index+1]*0x100000000);
        forceBuffers[atom+2*paddedNumAtoms] -= (long long) (grads[3*index+2]*0x100000000);
    }
}

